/*
 * Generates an array of random numbers
 * Reference: sortGPU.cu https://www.olcf.ornl.gov/tutorials/openacc-interoperability-ii/
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

// Fill d_buffer with num random numbers
// If you only need to generate on set of numbers and fill the array once,
// use this function. If you want to fill the array over and over again,
// use the other functions given below.
//
extern "C" void fill_rand(float *d_buffer, int num, void *stream, unsigned long long seed)
{
  hiprandGenerator_t gen;
  int status = HIPRAND_STATUS_SUCCESS;

  // Create generator
  status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

  // Set CUDA stream
  status |= hiprandSetStream(gen, (hipStream_t)stream);

  // Set seed
  status |= hiprandSetPseudoRandomGeneratorSeed(gen, seed);

  // Generate num random numbers
  // From documentation:
  //The curandGenerateUniform() function is used to generate uniformly
  // distributed floating point values between 0.0 and 1.0, 
  // where 0.0 is excluded and 1.0 is included.
  status |= hiprandGenerateUniform(gen, d_buffer, num);

  // Cleanup generator
  status |= hiprandDestroyGenerator(gen);

  if (status != HIPRAND_STATUS_SUCCESS) {
      printf ("hiprand failure!\n");
      exit (EXIT_FAILURE);
  }
}


//
// Set up a CUDA random number generator and return it.
//
extern "C" hiprandGenerator_t setup_prng(void *stream, unsigned long long seed) {
  hiprandGenerator_t gen;
  int status = HIPRAND_STATUS_SUCCESS;

  // Create generator
  status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

  // Set CUDA stream
  status |= hiprandSetStream(gen, (hipStream_t)stream);

  // Set seed
  status |= hiprandSetPseudoRandomGeneratorSeed(gen, seed);

  if (status != HIPRAND_STATUS_SUCCESS) {
      printf ("hiprand failure!\n");
      exit (EXIT_FAILURE);
  }

  return gen;
}

//
// Place a set of random numbers between 0.0 and 1.0 in an array d_buffer.
// This is designed so that with one generator (gen), this function can
// be called multiple times as needed to get a new set of random numbers
// for an iteration of a simulation, for example.
//
extern "C" void gen_rand_nums(hiprandGenerator_t gen, float *d_buffer, int num, void *stream) {
  int status = HIPRAND_STATUS_SUCCESS;

  // Generate num random numbers
  // From documentation:
  //The curandGenerateUniform() function is used to generate uniformly
  // distributed floating point values between 0.0 and 1.0, 
  // where 0.0 is excluded and 1.0 is included.
  status |= hiprandGenerateUniform(gen, d_buffer, num);

  if (status != HIPRAND_STATUS_SUCCESS) {
      printf ("hiprand failure!\n");
      exit (EXIT_FAILURE);
  }
}

//
// Remove the CUDA random number generator when finished with it.
//
extern "C" void rand_cleanup( hiprandGenerator_t gen ) {
  int status = HIPRAND_STATUS_SUCCESS;

  // Cleanup generator
  status |= hiprandDestroyGenerator(gen);

  if (status != HIPRAND_STATUS_SUCCESS) {
      printf ("hiprand failure!\n");
      exit (EXIT_FAILURE);
  }
}